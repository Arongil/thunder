#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
//#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcf;
template<int M_BLOCK, int N_BLOCK>
struct matmul_layout {
    using  base_tile      = st_bf<64, 64>;  // shared tile of size 64 by 64
    using  global_layout  = gl<bf16, 1, 1, -1, -1, base_tile>;
    struct globals        { global_layout A, B, C; };
    struct input_block    { base_tile a[M_BLOCK], b[N_BLOCK]; };  // outer product input
    struct finish_block   { base_tile c[M_BLOCK][N_BLOCK]; };     // outer product result
    struct common_state   { int2 coord; };
    struct consumer_state { rt_fl<16, N_BLOCK*base_tile::cols> accum; }; // register_tile of size 16 by N_BLOCK * however many columns we're doing
};
template<int _M_BLOCK=2, int _N_BLOCK=4, int _SUPER_M=12>
struct matmul_template {
    static constexpr int M_BLOCK = _M_BLOCK, N_BLOCK = _N_BLOCK, SUPER_M = _SUPER_M;
    using layout    = matmul_layout<M_BLOCK, N_BLOCK>;
    using wide_tile = st_bf<64, 64*N_BLOCK>;
    static constexpr int NUM_CONSUMER_WARPS=M_BLOCK*4, INPUT_PIPE_STAGES=4, PRODUCER_BARRIER_ARRIVALS=1;
    // Helper functions
    template<bool PERISISTENT_GRID=true> __host__ static inline dim3 grid(int M, int N, int K) {
        return dim3(PERISISTENT_GRID ? 132 : M*N/(M_BLOCK*N_BLOCK*layout::base_tile::num_elements));
    }
      // ThunderKittens template functions
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        // This is finding the number of "super" big blocks it can calculate in parallel,
        // then doing the remaining rows one-by-one
        int Rblocks = args.globals.C.rows / (M_BLOCK*64), Cblocks = args.globals.C.cols / (N_BLOCK*64);
        int super_rows = (Rblocks/SUPER_M)*SUPER_M,
            final_rows = Rblocks - super_rows,
            super_repeat = SUPER_M*Cblocks;
        int task_id = args.task_iter*gridDim.x + blockIdx.x;
        if (task_id < super_rows * Cblocks)
            args.common.coord = { SUPER_M*(task_id/super_repeat) + task_id%SUPER_M,
                           (task_id%super_repeat)/SUPER_M };
        else if (task_id < Rblocks*Cblocks) {
            int remainder_id = task_id - super_rows*Cblocks;
            args.common.coord = { super_rows + (remainder_id%final_rows), remainder_id/final_rows };
        }
        else { // Id is too high, no more work to do
            args.num_iters = -1;
            return;
        }
        args.num_iters = args.globals.A.cols/64;
        int id = warpgroup::groupid() == NUM_CONSUMER_WARPS/4 ? 0 : warpgroup::groupid(); // producer sets as 0
        args.common.coord = { args.common.coord.x*M_BLOCK + id, args.common.coord.y*N_BLOCK };
    }
    struct producer {
        __device__ static void setup(producer_setup_args<layout> args) {
            warpgroup::decrease_registers<40>(); // decrease registers for producers
        }
        __device__ static void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                for(int i = 0; i < M_BLOCK; i++)
                    tma::load_async(args.input.a[i], args.globals.A,
                                    {args.common.coord.x+i, args.iter}, args.inputs_arrived);
                for(int i = 0; i < N_BLOCK; i++)
                    tma::load_async(args.input.b[i], args.globals.B,
                                    {args.iter, args.common.coord.y+i}, args.inputs_arrived);
            }
        }
    };
    struct consumer {
        __device__ static void setup(consumer_setup_args<layout> args) {
            warpgroup::increase_registers<232>(); // increase registers for consumers
            zero(args.state.accum);
        }
        __device__ static void compute(consumer_compute_args<layout> args) {
            warpgroup::mma_AB(
                args.state.accum, // dest registers
                args.input.a[warpgroup::groupid()], // A matrix
                reinterpret_cast<wide_tile&>(args.input.b) // B matrix
            );
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.inputs_finished);
        }
        __device__ static void finish(consumer_finish_args<layout> args) {
            warpgroup::store(reinterpret_cast<wide_tile&>(args.finish.c[warpgroup::groupid()]), args.state.accum);
            warpgroup::sync(warpgroup::groupid()+4);
            if(warpgroup::warpid() == 0) for(int i = 0; i < N_BLOCK; i++) {
                tma::store_async(args.globals.C, args.finish.c[warpgroup::groupid()][i],
                                             {args.common.coord.x, args.common.coord.y+i});
                tma::store_async_read_wait(); // wait that store is finished before reusing finish memory
            }
            zero(args.state.accum);
            if(laneid() == 0) arrive(args.finish_finished);
		    args.globals.C[{0, 0, 0, 0}] = args.globals.A[{0,0,0,0}];
        }
    };
};

struct copy_layout {
    using base_tile     = st_bf<64, 64>;  // shared tile of size 64 by 64
    using global_layout = gl<bf16, 1, 1, -1, -1, base_tile>;
    struct globals { global_layout A, B, C; };
    struct input_block { base_tile a[1]; };
    struct common_state   { int2 coord; };
};
struct copy_template {
    using layout = copy_layout;
    static constexpr int NUM_CONSUMER_WARPS=4, INPUT_PIPE_STAGES=1, PRODUCER_BARRIER_ARRIVALS=1;
    // Helper functions
    __host__ static inline dim3 grid(int M, int N, int K) {
        return dim3(1, 1, 4096/64);
    }
    // ThunderKittens template functions
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        args.num_iters = args.globals.A.cols/64; 
    }
    struct producer {
        __device__ static void setup(producer_setup_args<layout> args) {
        }
        __device__ static void load(producer_load_args<layout> args) {
            /*
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                tma::load_async(args.input.a[0], args.globals.A, {0, args.iter}, args.inputs_arrived);
            }
            */
        }
    };
    struct consumer {
        __device__ static void setup(consumer_setup_args<layout> args) {
        }
        __device__ static void compute(consumer_compute_args<layout> args) {
            /*
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.inputs_finished);
            */
        }
        __device__ static void finish(consumer_finish_args<layout> args) {
            //warpgroup::store(args.globals.C[warpgroup::groupid()], args.finish.a[warpgroup::groupid()]);
            //if(laneid() == 0) arrive(args.finish_finished);
        }
    };
};

constexpr bool NCU = false;
#include <iostream>
#include <random>
#include <hip/hip_bf16.h>
#include <omp.h>

void cpu_gemm(float* a, float* b, float* c, int M, int N, int K) {
    #pragma omp parallel for collapse(2) // otherwise the CPU version takes for everrrrrr
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += a[i * K + k] * b[k * N + j];
            }
            c[i * N + j] = sum;
        }
    }
}

template<typename mmt>
void matmul(bf16 *d_A, bf16 *d_B, bf16 *d_C, size_t M, size_t N, size_t K, dim3 grid, dim3 block) {
    using global_layout = typename mmt::layout::global_layout;
    using globals  = typename mmt::layout::globals;
    global_layout Ag{d_A, nullptr, nullptr, M, K};
    global_layout Bg{d_B, nullptr, nullptr, K, N};
    global_layout Cg{d_C, nullptr, nullptr, M, N};
    globals G{Ag, Bg, Cg};
    prototype::lcf::kernel<mmt><<<grid, block, MAX_SHARED_MEMORY-1024>>>(G);
}

//using mmt = typename ::matmul_template<2,4,8>;
using mmt = typename ::copy_template;
using global_layout = typename mmt::layout::global_layout;
using globals = typename mmt::layout::globals;

/*
__global__ void one_kernel(const globals g) {
    if (blockIdx.x < g.C.batch && blockIdx.y < g.C.depth && blockIdx.z < g.C.rows && threadIdx.x < g.C.cols)
	    g.C[{blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x}] = g.A[{blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x}];
}
*/

void symmul4096_4096(globals g) {
	const size_t N = 4096;
	const size_t K = 4096;
    dim3 grid(mmt::grid(N, N, K));
    dim3 block(prototype::detail::NUM_THREADS_v<mmt>);

    // Create the layouts with explicit batch and depth values
    /*
    global_layout Ag{g.A.raw_ptr, nullptr, nullptr, N, K};  // batch=1, depth=1, rows=N, cols=K
    global_layout Bg{g.B.raw_ptr, nullptr, nullptr, K, N};  // batch=1, depth=1, rows=K, cols=N
    global_layout Cg{g.C.raw_ptr, nullptr, nullptr, N, N};  // batch=1, depth=1, rows=N, cols=N

    globals G{Ag, Bg, Cg};

    static_assert(Ag.batch == 1 && Ag.depth == 1);
    static_assert(Bg.batch == 1 && Bg.depth == 1);
    static_assert(Cg.batch == 1 && Cg.depth == 1);
    assert(Ag.batch == 1 && Ag.depth == 1);
    assert(Bg.batch == 1 && Bg.depth == 1);
    assert(Cg.batch == 1 && Cg.depth == 1);
    */

    // Print detailed information about the input matrices
    printf("\n=== Matrix Properties ===\n");
    printf("Matrix A:\n");
    printf("  Raw pointer: %p\n", g.A.raw_ptr);
    printf("  Batch: %d\n", g.A.batch);
    printf("  Depth: %d\n", g.A.depth); 
    printf("  Rows: %zu\n", g.A.rows);
    printf("  Cols: %zu\n", g.A.cols);

    printf("\nMatrix B:\n");
    printf("  Raw pointer: %p\n", g.B.raw_ptr);
    printf("  Batch: %d\n", g.B.batch);
    printf("  Depth: %d\n", g.B.depth);
    printf("  Rows: %zu\n", g.B.rows);
    printf("  Cols: %zu\n", g.B.cols);

    printf("\nMatrix C:\n"); 
    printf("  Raw pointer: %p\n", g.C.raw_ptr);
    printf("  Batch: %d\n", g.C.batch);
    printf("  Depth: %d\n", g.C.depth);
    printf("  Rows: %zu\n", g.C.rows);
    printf("  Cols: %zu\n", g.C.cols);

    printf("\nGrid dimensions: (%d, %d, %d)\n", grid.x, grid.y, grid.z);
    printf("Block dimensions: (%d, %d, %d)\n", block.x, block.y, block.z);
    printf("Shared memory size: %d\n", MAX_SHARED_MEMORY-1024);
    printf("======================\n\n");

    //one_kernel<<<grid, block>>>(G);
    //prototype::lcf::kernel<matmul_template<2,4,8>><<<grid, block, MAX_SHARED_MEMORY-1024>>>(g);
    prototype::lcf::kernel<mmt><<<grid, block, MAX_SHARED_MEMORY-1024>>>(g);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }
    
    // Wait for kernel to finish and check for errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel execution failed: %s\n", hipGetErrorString(err));
        return;
    }
    
    printf("Kernel completed successfully\n");
}

/*
void symmul4096_16384(globals g) {
	dim3 grid(mmt::grid(4096, 4096, 16384));
	dim3 block(prototype::detail::NUM_THREADS_v<mmt>);
    prototype::lcf::kernel<mmt><<<grid, block, MAX_SHARED_MEMORY-1024>>>(g);
}
*/

/*
PYBIND11_MODULE(symmul, m) {
    m.doc() = "ThunderKittens symmetric matmul module";
    BIND_FUNCTION(m, "symmul4096_4096", symmul4096_4096, globals, A, B, C);
//BIND_FUNCTION(m, "symmul4096_16384", symmul4096_16384, globals);
}
*/

////////////////////////////////////////////////////////////
// Benchmarking
////////////////////////////////////////////////////////////

template<typename mmt>
int run_benchmark(size_t M, size_t N, size_t K) {
    hipError_t cudaStatus;

    std::cout << "--------------------  M=" << M << " N=" << N << " K=" << K << "  --------------------\n";
    std::cout << "Block size: " << mmt::M_BLOCK*64 << "x" << mmt::N_BLOCK*64 << "\n";

    // Allocate host memory
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];
    float *h_C_ref = new float[M * N];

    std::cout << "Allocated host memory" << std::endl;

    // Initialize random number generator
    std::random_device rd;
    std::mt19937 gen(42);
    std::uniform_real_distribution<> dis(-0.5, 0.5);

    // Initialize matrices with random values
    for (int i = 0; i < M * K; ++i) h_A[i] = dis(gen);
    for (int i = 0; i < K * N; ++i) h_B[i] = dis(gen);

    std::cout << "Initialized matrices" << std::endl;

    // Perform CPU matrix multiplication for reference
    if(true) cpu_gemm(h_A, h_B, h_C_ref, M, N, K);

    std::cout << "Performed CPU matrix multiplication" << std::endl;

    // Allocate device memory
    __hip_bfloat16 *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M*K*sizeof(__hip_bfloat16));
    hipMalloc(&d_B, K*N*sizeof(__hip_bfloat16));
    hipMalloc(&d_C, M*N*sizeof(__hip_bfloat16));

    // Check for CUDA errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
        // Optionally, you might want to exit the program or handle the error in some way
        return -1;
    }

    std::cout << "Allocated device memory" << std::endl;

    // Convert to __hip_bfloat16 and copy to device
    __hip_bfloat16 *h_A_bf16 = new __hip_bfloat16[M * K];
    __hip_bfloat16 *h_B_bf16 = new __hip_bfloat16[K * N];
    for (int i = 0; i < M * K; ++i) h_A_bf16[i] = __float2bfloat16(h_A[i]);
    for (int i = 0; i < K * N; ++i) h_B_bf16[i] = __float2bfloat16(h_B[i]);

    hipMemcpy(d_A, h_A_bf16, M*K*2, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_bf16, K*N*2, hipMemcpyHostToDevice);

    std::cout << "Copied matrices to device" << std::endl;

    unsigned long mem_size = MAX_SHARED_MEMORY - 1024;
    hipFuncSetAttribute(reinterpret_cast<const void*>(prototype::lcf::kernel<mmt>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

    // Launch kernel
    dim3 grid(mmt::grid(M, N, K));
    dim3 block(kittens::prototype::detail::NUM_THREADS_v<mmt>);
    std::cout << "Launching warmup kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ")\n";
    for(int i = 0; i < (NCU ? 0 : 2); i++) { // warmup
        matmul<mmt>(d_A, d_B, d_C, M, N, K, grid, block);
    }

    // Start timing
    hipDeviceSynchronize();
    std::cout << "Launching kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ")\n";
    auto start = std::chrono::high_resolution_clock::now();

    constexpr int ITERS = (NCU ? 1 : 10);
    for(int i = 0; i < ITERS; i++) {
        matmul<mmt>(d_A, d_B, d_C, M, N, K, grid, block);
    }
    hipDeviceSynchronize();

    // End timing
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate duration
    std::chrono::duration<double> diff = end - start;
    double useconds = diff.count() * 1e6 / ITERS;

    // Calculate TFLOPs
    double flops = double(2.0) * M * N * K; // 2 FLOPs per multiply-add
    double tflops = (flops / useconds) / 1e6;

    std::cout << "Avg Kernel execution time: " << useconds << " us\n";
    std::cout << "Achieved performance: " << tflops << " TFLOPs\n";
    
    // Check for CUDA errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
        // Optionally, you might want to exit the program or handle the error in some way
        return -1;
    }

    // Copy result back to host
    __hip_bfloat16 *h_C_bf16 = new __hip_bfloat16[M * N];
    hipMemcpy(h_C_bf16, d_C, M*N*2, hipMemcpyDeviceToHost);

    std::cout << "Copied result back to host" << std::endl;

    // Convert result back to float for comparison
    for (int i = 0; i < M * N; ++i) h_C[i] = __bfloat162float(h_C_bf16[i]);

    std::cout << "Converted result back to float" << std::endl;

    // Check result
    float max_error = 0.0f;
    int error_count = 0;
    for (int i = 0; i < M * N; ++i) {
        float error = std::abs(h_C[i] - h_C_ref[i]);
        if(error > 1.0) { // large because of bf16 vs fp32 numerics
            if(error_count < 20) std::cout << "Error at row " << i / N << " col " << i % N << ": " << h_C[i] << " != " << h_C_ref[i] << " (ref)" << std::endl;
            else if(error_count == 21) std::cout << "Too many errors to show them all.\n";
            error_count++;
        }
        max_error = std::max(max_error, error);
    }

    std::cout << "Max error: " << max_error << std::endl;
    std::cout << "Error count: " << error_count << std::endl;

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_ref;
    delete[] h_A_bf16;
    delete[] h_B_bf16;
    delete[] h_C_bf16;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

int main() {
    int N;
    N = 1024;
    run_benchmark<matmul_template<2,4,8>>(N, N, N);
    // N = 3072;
    // run_benchmark<matmul_template<2,4,8>>(N, N, N);
    // run_benchmark<matmul_template<3,3,8>>(N, N, N);
    // N = 4096;
    // run_benchmark<matmul_template<2,4,8>>(N, N, N);
    // N = 6144;
    // run_benchmark<matmul_template<2,4,8>>(N, N, N);
    // run_benchmark<matmul_template<3,3,8>>(N, N, N);
    // N = 8192;
    // run_benchmark<matmul_template<2,4,8>>(N, N, N);
    // N = 12288;
    // run_benchmark<matmul_template<2,4,8>>(N, N, N);
    // run_benchmark<matmul_template<3,3,8>>(N, N, N);
    // N = 16384;
    // run_benchmark<matmul_template<2,4,8>>(N, N, N);
    // run_benchmark<matmul_template<2,4,12>>(N, N, N);
    // run_benchmark<matmul_template<3,3,12>>(192*12, 192*11, 8192);
    // run_benchmark<matmul_template<2,4,11>>(128*22, 256* 6, 8192);
    // run_benchmark<matmul_template<2,4,1>>(128 * 132, 256, 256);
    // run_benchmark<matmul_template<2,4,1>>(128 * 133, 256, 256);
    // run_benchmark<matmul_template<2,4,1>>(16384, 16384, 16384);
    // run_benchmark<matmul_template<2,4,8>>(16384, 16384, 16384);
    // run_benchmark<matmul_template<2,4,12>>(16384, 16384, 16384);
    // run_benchmark<matmul_template<2,4,128>>(16384, 16384, 16384);
    // run_benchmark<matmul_template<3,3,12>>(192*22, 192*6*2, 8192);
    // run_benchmark<matmul_template<3,3,12>>(192*22, 192*6*2, 16384);
    // run_benchmark<matmul_template<2,4,11>>(128*22*2, 256* 6*2, 8192);
    // run_benchmark<matmul_template<3,3,12>>(192*12*2, 192*11*2, 8192*2);
    // run_benchmark<matmul_template<2,4,11>>(128*22*2, 256* 6*2, 8192*2);
    return 0;
}