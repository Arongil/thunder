#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcf;
template<int M_BLOCK, int N_BLOCK>
struct symmul_layout {
    using  base_tile      = st_bf<64, 64>;  // shared tile of size 64 by 64
    using  global_layout  = gl<bf16, 1, 1, -1, -1, base_tile>;
    struct globals        { global_layout A, B, C; };
    struct input_block    { base_tile a[M_BLOCK], b[N_BLOCK]; };  // outer product input
    struct finish_block   { base_tile c[M_BLOCK][N_BLOCK]; };     // outer product result
    struct common_state   { int2 coord; };
    struct consumer_state { rt_fl<16, N_BLOCK*base_tile::cols> accum; rt_bf<16, N_BLOCK*base_tile::cols> accum_bf16; rt_bf<N_BLOCK*base_tile::cols, 16> accum_transposed; }; // register_tile of size 16 by N_BLOCK * however many columns we're doing
};
template<int _M_BLOCK=2, int _N_BLOCK=4, int _SUPER_M=12>
struct symmul_template {
    static constexpr int M_BLOCK = _M_BLOCK, N_BLOCK = _N_BLOCK, SUPER_M = _SUPER_M;
    using layout    = symmul_layout<M_BLOCK, N_BLOCK>;
    using wide_tile = st_bf<64, 64*N_BLOCK>;  // 64 rows by 64*N_BLOCK columns
    using tall_tile = st_bf<64*N_BLOCK, 16>;  // 64*N_BLOCK rows by 16 columns
    static constexpr int NUM_CONSUMER_WARPS=M_BLOCK*4, INPUT_PIPE_STAGES=4, PRODUCER_BARRIER_ARRIVALS=1;
    // Helper functions
    template<bool PERISISTENT_GRID=true> __host__ static inline dim3 grid(int M, int N) {
        return dim3(PERISISTENT_GRID ? 132 : M*N/(M_BLOCK*N_BLOCK*layout::base_tile::num_elements));
    }
      // ThunderKittens template functions
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        int Rblocks = args.globals.C.rows / (M_BLOCK*64), Cblocks = args.globals.C.cols / (N_BLOCK*64);
        int super_rows = (Rblocks/SUPER_M)*SUPER_M,
            final_rows = Rblocks - super_rows,
            super_repeat = SUPER_M*Cblocks;
        int task_id = args.task_iter*gridDim.x + blockIdx.x;
        if (task_id < super_rows * Cblocks)
            args.common.coord = { SUPER_M*(task_id/super_repeat) + task_id%SUPER_M,
                           (task_id%super_repeat)/SUPER_M };
                           // ^ row, col ... of the block
        else if (task_id < Rblocks*Cblocks) {
            int remainder_id = task_id - super_rows*Cblocks;
            args.common.coord = { super_rows + (remainder_id%final_rows), remainder_id/final_rows };
        }
        else { // Id is too high, no more work to do
            args.num_iters = -1;
            return;
        }

        args.num_iters = args.globals.A.cols/64;
        int id = warpgroup::groupid() == NUM_CONSUMER_WARPS/4 ? 0 : warpgroup::groupid(); // producer sets as 0
        args.common.coord = { args.common.coord.x*M_BLOCK + id, args.common.coord.y*N_BLOCK };

        // Skip upper triangular blocks
        /*
        if (args.common.coord.x < args.common.coord.y) {
            args.num_iters = -1;
            return;
        }
        */
    }
    struct producer {
        __device__ static void setup(producer_setup_args<layout> args) {
            warpgroup::decrease_registers<40>(); // decrease registers for producers
        }
        __device__ static void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                // load the outer product input information
                for(int i = 0; i < M_BLOCK; i++)
                    tma::load_async(args.input.a[i], args.globals.A,
                                    {args.common.coord.x+i, args.iter}, args.inputs_arrived);
                                    //                      ^ task_iter goes from 0 to however much we need to traverse the K dimension
                for(int i = 0; i < N_BLOCK; i++)
                    tma::load_async(args.input.b[i], args.globals.B,
                                    {args.iter, args.common.coord.y+i}, args.inputs_arrived);
            }
        }
    };
    struct consumer {
        __device__ static void setup(consumer_setup_args<layout> args) {
            warpgroup::increase_registers<232>(); // increase registers for consumers
            zero(args.state.accum);
            zero(args.state.accum_transposed);
        }
        __device__ static void compute(consumer_compute_args<layout> args) {
            warpgroup::mma_AB(
                args.state.accum, // dest registers
                args.input.a[warpgroup::groupid()], // A matrix
                reinterpret_cast<wide_tile&>(args.input.b) // B matrix
            );
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.inputs_finished);
        }
        __device__ static void finish(consumer_finish_args<layout> args) {
            // Store the lower triangular part
            warpgroup::store(reinterpret_cast<wide_tile&>(args.finish.c[warpgroup::groupid()]), args.state.accum);
            warpgroup::sync(warpgroup::groupid()+4);
            if(warpgroup::warpid() == 0) {
                for(int i = 0; i < N_BLOCK; i++) {
                    // store lower triangular part
                    tma::store_async(args.globals.C, args.finish.c[warpgroup::groupid()][i],
                                                {args.common.coord.x, args.common.coord.y+i});
                    tma::store_async_read_wait(); // wait that store is finished before reusing finish memory
                }
            }

            // store the upper triangular part -- first convert to bf16 so that we can tranpose
            kittens::copy(args.state.accum_bf16, args.state.accum);
            kittens::transpose_sep(args.state.accum_transposed, args.state.accum_bf16);
            auto &c_transposed = reinterpret_cast<layout::base_tile(&)[N_BLOCK][M_BLOCK]>(args.finish.c);

            warpgroup::store(reinterpret_cast<tall_tile&>(c_transposed[warpgroup::groupid()]), args.state.accum_transposed);
            warpgroup::sync(warpgroup::groupid()+4);
            if(warpgroup::warpid() == 0) {
                for(int i = 0; i < N_BLOCK; i++) {
                    tma::store_async(args.globals.C, c_transposed[i][warpgroup::groupid()],
                                                {args.common.coord.y+i, args.common.coord.x});
                    tma::store_async_read_wait(); // wait that store is finished before reusing finish memory
                }
            }

            zero(args.state.accum);
            zero(args.state.accum_transposed);
            if(laneid() == 0) arrive(args.finish_finished);
        }
    };
};


// ORIGINAL MATMUL TEMPLATE //
template<int M_BLOCK, int N_BLOCK>
struct matmul_layout {
    using  base_tile      = st_bf<64, 64>;
    using  global_layout  = gl<bf16, 1, 1, -1, -1, base_tile>;
    struct globals        { global_layout A, B, C; };
    struct input_block    { base_tile a[M_BLOCK], b[N_BLOCK]; };
    struct finish_block   { base_tile c[M_BLOCK][N_BLOCK]; };
    struct common_state   { int2 coord; };
    struct consumer_state { rt_fl<16, N_BLOCK*base_tile::cols> accum; };
};
template<int _M_BLOCK=2, int _N_BLOCK=4, int _SUPER_M=12>
struct matmul_template {
    static constexpr int M_BLOCK = _M_BLOCK, N_BLOCK = _N_BLOCK, SUPER_M = _SUPER_M;
    using layout    = matmul_layout<M_BLOCK, N_BLOCK>;
    using wide_tile = st_bf<64, 64*N_BLOCK>;
    static constexpr int NUM_CONSUMER_WARPS=M_BLOCK*4, INPUT_PIPE_STAGES=4, PRODUCER_BARRIER_ARRIVALS=1;
    // Helper functions
    template<bool PERISISTENT_GRID=true> __host__ static inline dim3 grid(int M, int N, int K) {
        return dim3(PERISISTENT_GRID ? 132 : M*N/(M_BLOCK*N_BLOCK*layout::base_tile::num_elements));
    }
      // ThunderKittens template functions
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        // This is finding the number of "super" big blocks it can calculate in parallel,
        // then doing the remaining rows one-by-one
        int Rblocks = args.globals.C.rows / (M_BLOCK*64), Cblocks = args.globals.C.cols / (N_BLOCK*64);
        int super_rows = (Rblocks/SUPER_M)*SUPER_M,
            final_rows = Rblocks - super_rows,
            super_repeat = SUPER_M*Cblocks;
        int task_id = args.task_iter*gridDim.x + blockIdx.x;
        if (task_id < super_rows * Cblocks)
            args.common.coord = { SUPER_M*(task_id/super_repeat) + task_id%SUPER_M,
                           (task_id%super_repeat)/SUPER_M };
        else if (task_id < Rblocks*Cblocks) {
            int remainder_id = task_id - super_rows*Cblocks;
            args.common.coord = { super_rows + (remainder_id%final_rows), remainder_id/final_rows };
        }
        else { // Id is too high, no more work to do
            args.num_iters = -1;
            return;
        }
        args.num_iters = args.globals.A.cols/64;
        int id = warpgroup::groupid() == NUM_CONSUMER_WARPS/4 ? 0 : warpgroup::groupid(); // producer sets as 0
        args.common.coord = { args.common.coord.x*M_BLOCK + id, args.common.coord.y*N_BLOCK };
    }
    struct producer {
        __device__ static void setup(producer_setup_args<layout> args) {
            warpgroup::decrease_registers<40>(); // decrease registers for producers
        }
        __device__ static void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                for(int i = 0; i < M_BLOCK; i++)
                    tma::load_async(args.input.a[i], args.globals.A,
                                    {args.common.coord.x+i, args.iter}, args.inputs_arrived);
                for(int i = 0; i < N_BLOCK; i++)
                    tma::load_async(args.input.b[i], args.globals.B,
                                    {args.iter, args.common.coord.y+i}, args.inputs_arrived);
            }
        }
    };
    struct consumer {
        __device__ static void setup(consumer_setup_args<layout> args) {
            warpgroup::increase_registers<232>(); // increase registers for consumers
            zero(args.state.accum);
        }
        __device__ static void compute(consumer_compute_args<layout> args) {
            warpgroup::mma_AB(
                args.state.accum, // dest registers
                args.input.a[warpgroup::groupid()], // A matrix
                reinterpret_cast<wide_tile&>(args.input.b) // B matrix
            );
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.inputs_finished);
        }
        __device__ static void finish(consumer_finish_args<layout> args) {
            warpgroup::store(reinterpret_cast<wide_tile&>(args.finish.c[warpgroup::groupid()]), args.state.accum);
            warpgroup::sync(warpgroup::groupid()+4);
            if(warpgroup::warpid() == 0) for(int i = 0; i < N_BLOCK; i++) {
                tma::store_async(args.globals.C, args.finish.c[warpgroup::groupid()][i],
                                             {args.common.coord.x, args.common.coord.y+i});
                tma::store_async_read_wait(); // wait that store is finished before reusing finish memory
            }
            zero(args.state.accum);
            if(laneid() == 0) arrive(args.finish_finished);
        }
    };
};
// END ORIGINAL MATMUL TEMPLATE //

#include <hip/hip_bf16.h>

template<typename mmt>
void matmul(bf16 *d_A, bf16 *d_B, bf16 *d_C, size_t M, size_t N, size_t K, dim3 grid, dim3 block) {
    using global_layout = typename mmt::layout::global_layout;
    using globals  = typename mmt::layout::globals;
    global_layout Ag{d_A, nullptr, nullptr, M, K};
    global_layout Bg{d_B, nullptr, nullptr, K, N};
    global_layout Cg{d_C, nullptr, nullptr, M, N};
    globals G{Ag, Bg, Cg};
    prototype::lcf::kernel<mmt><<<grid, block, MAX_SHARED_MEMORY-1024>>>(G);
}

using mmt = matmul_template<2,4,8>;
using smt = symmul_template<2,4,8>;
using mmt_globals = typename mmt::layout::globals;
using smt_globals = typename smt::layout::globals;

void matmul4096_4096(mmt_globals g) {
    const size_t N = 4096;
    const size_t M = 4096;
    const size_t K = 4096;
    dim3 grid(mmt::grid(M, N, K));
    dim3 block(prototype::detail::NUM_THREADS_v<mmt>);

    // Absolutely need to set the dynamic shared memory here!!!
    unsigned long mem_size = MAX_SHARED_MEMORY - 1024;
    hipFuncSetAttribute(reinterpret_cast<const void*>(prototype::lcf::kernel<mmt>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
    prototype::lcf::kernel<mmt><<<grid, block, MAX_SHARED_MEMORY-1024>>>(g);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }
}

void symmul4096_4096(smt_globals g) {
    const size_t N = 4096;
    const size_t K = 4096;
    dim3 grid(smt::grid(N, K));
    dim3 block(prototype::detail::NUM_THREADS_v<smt>);

    // Absolutely need to set the dynamic shared memory here!!!
    unsigned long mem_size = MAX_SHARED_MEMORY - 1024;
    hipFuncSetAttribute(reinterpret_cast<const void*>(prototype::lcf::kernel<smt>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
    prototype::lcf::kernel<smt><<<grid, block, MAX_SHARED_MEMORY-1024>>>(g);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }
}

PYBIND11_MODULE(symmul, m) {
    m.doc() = "ThunderKittens symmetric matmul module";
    BIND_FUNCTION(m, "matmul4096_4096", matmul4096_4096, mmt_globals, A, B, C);
    BIND_FUNCTION(m, "symmul4096_4096", symmul4096_4096, smt_globals, A, B, C);
}